#include "cracker.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include "sha1_kernel.cu"


void bytes_to_uint32_be(const uint8_t *bytes, uint32_t *out, int count)
{
  for (int i = 0; i < count; ++i)
  {
    out[i] = ((uint32_t)bytes[i * 4] << 24) |
             ((uint32_t)bytes[i * 4 + 1] << 16) |
             ((uint32_t)bytes[i * 4 + 2] << 8) | (uint32_t)bytes[i * 4 + 3];
  }
}
extern "C" int initCuda(GPUInfo *info)
{
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
  {
    fprintf(stderr, "CUDA error: No CUDA-enabled devices found\n");
    return -1;
  }
  int dev = 0;
  hipSetDevice(dev);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  strncpy(info->name, deviceProp.name, 255);
  info->name[255] = '\0';
  info->multiProcessorCount = deviceProp.multiProcessorCount;
  info->cudaMajor = deviceProp.major;
  info->cudaMinor = deviceProp.minor;
  info->totalGlobalMem = deviceProp.totalGlobalMem;
  return 0;
}


extern "C" int searchOnGpu(const uint8_t* target_hash_bytes, uint64_t start_index, uint64_t count, char* result_str, int length, const char* alphabet, int alphabet_len) {
    if (length > MAX_CRACK_LENGTH) return -1;
    if (alphabet_len > MAX_ALPHABET_SIZE) return -1;

    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(d_alphabet), alphabet, alphabet_len * sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpyToSymbol failed: %s\n", hipGetErrorString(err));
        return -1;
    }

    uint32_t target_hash_u32[5];
    bytes_to_uint32_be(target_hash_bytes, target_hash_u32, 5);
    uint32_t* d_target_hash;
    volatile uint32_t* d_found_flag;
    char* d_result_str;
    uint32_t h_found_flag = 0;
    err = hipMalloc((void**)&d_target_hash, 5 * sizeof(uint32_t)); if (err != hipSuccess) return -1;
    err = hipMalloc((void**)&d_found_flag, sizeof(uint32_t)); if (err != hipSuccess) return -1;
    err = hipMalloc((void**)&d_result_str, (MAX_CRACK_LENGTH + 1) * sizeof(char)); if (err != hipSuccess) return -1;
    err = hipMemcpy(d_target_hash, target_hash_u32, 5 * sizeof(uint32_t), hipMemcpyHostToDevice); if (err != hipSuccess) return -1;
    err = hipMemcpy((void*)d_found_flag, &h_found_flag, sizeof(uint32_t), hipMemcpyHostToDevice); if (err != hipSuccess) return -1;
    int threadsPerBlock = 256;
    dim3 blocksPerGrid((unsigned int)((count + threadsPerBlock - 1) / threadsPerBlock), 1, 1);
    int maxGridSize;
    hipDeviceGetAttribute(&maxGridSize, hipDeviceAttributeMaxGridDimX, 0);
    if (blocksPerGrid.x > maxGridSize) { blocksPerGrid.x = maxGridSize; }
    sha1_search_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_target_hash, start_index, d_found_flag, d_result_str, length, alphabet_len);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        return -1;
    }
    hipDeviceSynchronize();
    hipMemcpy(&h_found_flag, (const void*)d_found_flag, sizeof(uint32_t), hipMemcpyDeviceToHost);
    int result = 0;
    if (h_found_flag != 0) {
        hipMemcpy(result_str, d_result_str, (length + 1) * sizeof(char), hipMemcpyDeviceToHost);
        result = 1;
    }
    hipFree(d_target_hash);
    hipFree((void*)d_found_flag);
    hipFree(d_result_str);
    return result;
}

extern "C" void cleanupCuda() {
    hipDeviceReset();
}